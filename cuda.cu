/* 
 * Ускорение программы с помощью CUDA.
 * Вариант: 7
 * Бизнес логика: Умножение матрицы на вектор
 * 
 * Считывание данных происходит из файла.
 * Программа выполняет бизнес-логику и записывает результат в выходной файл.
 * В конце файла с результатами сохраняется информация о времени выполнения вычислений 
 * и размере обработанных данных.
 *
 * Запуск: nvcc cuda.cu utils.c -o cuda.out && \
$PWD/cuda.out ./test_data/1mb ./results/cuda/1mb
 */

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define DEBUG 0
#define LOG 1

void read_matrix(FILE *input_file, int **matrix, long matrix_size);

void read_vector(FILE *input_file, int *vector, long vector_length);

void print_vector(const int *vector, long vector_length);

__global__
void calc_answer(int **matrix, const int *vector, int *answer, long vector_length);

void save_answer(FILE *output_file, const int *answer, long answer_length);

int main(int argc, char *argv[], char *argp[]) {

    char *input_file_name;
    char *output_file_name;

    if (argc < 3) {
        input_file_name = "input_file";
        output_file_name = "output_file";
    } else {
        input_file_name = argv[1];
        output_file_name = argv[2];
    }

    if (LOG) printf("input file name: %s,\noutput file name: %s.\n\n", input_file_name, output_file_name);

    FILE *input_file = NULL;
    input_file = fopen(input_file_name, "r+");
    if (input_file == NULL) {
        printf("input file not found!");
        return -1;
    }

    long matrix_size;
    fscanf(input_file, "%ld", &matrix_size);
    if (LOG) printf("matrix_size: %ld \n", matrix_size);

    int **matrix = (int **) calloc(matrix_size, sizeof(int *));
    for (long i = 0; i < matrix_size; i++) matrix[i] = (int *) calloc(matrix_size, sizeof(int));
    read_matrix(input_file, matrix, matrix_size);

    long vector_length = matrix_size;
    int *vector = (int *) calloc(vector_length, sizeof(int));
    read_vector(input_file, vector, vector_length);

    int *answer = (int *) calloc(vector_length, sizeof(int));

    int **dev_matrix;
    int *dev_vector, *dev_answer;

    hipMalloc((void **) &dev_matrix, sizeof(int) * matrix_size * matrix_size);
    hipMalloc((void **) &dev_vector, sizeof(int) * vector_length);
    hipMalloc((void **) &dev_answer, sizeof(int) * vector_length);

    hipMemcpy(dev_matrix, matrix, sizeof(int) * matrix_size * matrix_size, hipMemcpyHostToDevice);
    hipMemcpy(dev_vector, vector, sizeof(int) * vector_length, hipMemcpyHostToDevice);

    clock_t begin = clock();
    calc_answer<<<matrix_size / 256 + 1, 256 >>>(dev_matrix, dev_vector, dev_answer, vector_length);
    clock_t end = clock();

    double time_spent_in_sec = (double) (end - begin) / CLOCKS_PER_SEC;

    hipMemcpy(answer, dev_answer, sizeof(int) * vector_length, hipMemcpyDeviceToHost);

    hipFree(dev_matrix);
    hipFree(dev_vector);
    hipFree(dev_answer);

    FILE *output_file = NULL;
    output_file = fopen(output_file_name, "w+");

    if (output_file == NULL) {
        printf("output file not found!");
        return -1;
    }

    save_answer(output_file, answer, vector_length);

    fprintf(output_file, "time: %f\n", time_spent_in_sec);
    if (LOG) printf("time_spent_in_sec: %f\n", time_spent_in_sec);

    long size_of_input_data = (long) ((matrix_size * matrix_size + matrix_size) * sizeof(int));
    double size_of_input_data_in_mb = (double) size_of_input_data / 1024 / 1024;
    fprintf(output_file, "size: %f\n", size_of_input_data_in_mb);
    if (LOG) {
        printf("matrix_size: %ld\n", matrix_size);
        printf("size_of_input_data: %ld\n", size_of_input_data);
        printf("size_of_input_data_in_mb: %f\n\n", size_of_input_data_in_mb);
    }

    fclose(input_file);
    fclose(output_file);

    for (int i = 0; i < matrix_size; i++) free(matrix[i]);
    free(matrix);
    free(vector);
    free(answer);

    return 0;
}

void read_matrix(FILE *input_file, int **matrix, long matrix_size) {
    if (DEBUG) printf("read_matrix:\n");
    for (long i = 0; i < matrix_size; i++) {
        for (long j = 0; j < matrix_size; j++) {
            fscanf(input_file, "%d", &matrix[i][j]);
            if (DEBUG) printf("%d ", matrix[i][j]);
        }
        if (DEBUG) printf("\n");
    }

    if (DEBUG) printf("\n");
}

void read_vector(FILE *input_file, int *vector, long vector_length) {
    for (long i = 0; i < vector_length; i++) {
        fscanf(input_file, "%d", &vector[i]);
    }

    if (DEBUG) print_vector(vector, vector_length);
}

void print_vector(const int *vector, long vector_length) {
    printf("read_vector:\n");
    for (long i = 0; i < vector_length; i++) {
        printf("%d ", vector[i]);
    }
    printf("\n\n");
}

__global__
void calc_answer(int **matrix, const int *vector, int *answer, long vector_length) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = tid; i < vector_length; i += stride) {
        for (long j = 0; j < vector_length; j++) {
            answer[i] += matrix[j][i] * vector[j];
        }
    }
}

void save_answer(FILE *output_file, const int *answer, long answer_length) {
    if (DEBUG) printf("save_answer:\n");
    fprintf(output_file, "result:\n");

    for (long i = 0; i < answer_length; i++) {
        fprintf(output_file, "%d ", answer[i]);
        if (DEBUG) printf("%d ", answer[i]);
    }
    fprintf(output_file, "\n");
    if (DEBUG) printf("\n\n");
}
