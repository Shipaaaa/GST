/* 
 * Ускорение программы с помощью CUDA.
 * Вариант: 7
 * Бизнес логика: Умножение матрицы на вектор
 * 
 * Считывание данных происходит из файла.
 * Программа выполняет бизнес-логику и записывает результат в выходной файл.
 * В конце файла с результатами сохраняется информация о времени выполнения вычислений 
 * и размере обработанных данных.
 *
 * Запуск: nvcc cuda.cu -o cuda.out && \
$PWD/cuda.out ./test_data/1mb ./results/cuda/1mb
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define DEBUG 0
#define LOG 1

void read_matrix(FILE *input_file, int *matrix, long matrix_size);

void read_vector(FILE *input_file, int *vector, long vector_length);

void print_vector(const int *vector, long vector_length);

__global__
void calc_answer(const int *matrix, const int *vector, int *answer, long vector_length);

void save_answer(FILE *output_file, const int *answer, long answer_length);

int main(int argc, char *argv[], char *argp[]) {

    char *input_file_name;
    char *output_file_name;

    if (argc < 3) {
        input_file_name = "input_file";
        output_file_name = "output_file";
    } else {
        input_file_name = argv[1];
        output_file_name = argv[2];
    }

    if (LOG) printf("input file name: %s,\noutput file name: %s.\n\n", input_file_name, output_file_name);

    FILE *input_file = NULL;
    input_file = fopen(input_file_name, "r+");
    if (input_file == NULL) {
        printf("input file not found!");
        return -1;
    }

    long matrix_size;
    fscanf(input_file, "%ld", &matrix_size);
    if (LOG) printf("matrix_size: %ld \n", matrix_size);

    int *matrix = (int *) calloc(matrix_size * matrix_size, sizeof(int));
    read_matrix(input_file, matrix, matrix_size);

    long vector_length = matrix_size;
    int *vector = (int *) calloc(vector_length, sizeof(int));
    read_vector(input_file, vector, vector_length);

    int *answer = (int *) calloc(vector_length, sizeof(int));

    int *dev_matrix, *dev_vector, *dev_answer;

    int deviceId;
    int numberOfSMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

    hipMalloc(&dev_matrix, matrix_size * matrix_size * sizeof(int));
    hipMalloc(&dev_vector, vector_length * sizeof(int));
    hipMalloc(&dev_answer, vector_length * sizeof(int));

    hipMemcpy(dev_matrix, matrix, matrix_size * matrix_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_vector, vector, vector_length * sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t begin, end;
    float time_spent_in_sec;
    hipEventCreate(&begin);
    hipEventCreate(&end);

    int number_of_blocks = numberOfSMs * 32;
    int threads_per_block = 256;

    hipEventRecord(begin, 0);
    calc_answer<<<number_of_blocks, threads_per_block>>>(dev_matrix, dev_vector, dev_answer, vector_length);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);

    hipEventElapsedTime(&time_spent_in_sec, begin, end);
    time_spent_in_sec /= 1000;

    hipMemcpy(answer, dev_answer, vector_length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_matrix);
    hipFree(dev_vector);
    hipFree(dev_answer);

    FILE *output_file = NULL;
    output_file = fopen(output_file_name, "w+");

    if (output_file == NULL) {
        printf("output file not found!");
        return -1;
    }

    save_answer(output_file, answer, vector_length);

    fprintf(output_file, "time: %f\n", time_spent_in_sec);
    if (LOG) printf("time_spent_in_sec: %f\n", time_spent_in_sec);

    long size_of_input_data = (long) ((matrix_size * matrix_size + matrix_size) * sizeof(int));
    double size_of_input_data_in_mb = (double) size_of_input_data / 1024 / 1024;
    fprintf(output_file, "size: %f\n", size_of_input_data_in_mb);
    if (LOG) {
        printf("matrix_size: %ld\n", matrix_size);
        printf("size_of_input_data: %ld\n", size_of_input_data);
        printf("size_of_input_data_in_mb: %f\n\n", size_of_input_data_in_mb);
    }

    fclose(input_file);
    fclose(output_file);

    free(matrix);
    free(vector);
    free(answer);

    return 0;
}

void read_matrix(FILE *input_file, int *matrix, long matrix_size) {
    if (DEBUG) printf("read_matrix:\n");
    for (long i = 0; i < matrix_size; i++) {
        for (long j = 0; j < matrix_size; j++) {
            fscanf(input_file, "%d", &matrix[i * matrix_size + j]);
            if (DEBUG) printf("%d ", matrix[i * matrix_size + j]);
        }
        if (DEBUG) printf("\n");
    }

    if (DEBUG) printf("\n");
}

void read_vector(FILE *input_file, int *vector, long vector_length) {
    for (long i = 0; i < vector_length; i++) {
        fscanf(input_file, "%d", &vector[i]);
    }

    if (DEBUG) print_vector(vector, vector_length);
}

void print_vector(const int *vector, long vector_length) {
    printf("read_vector:\n");
    for (long i = 0; i < vector_length; i++) {
        printf("%d ", vector[i]);
    }
    printf("\n\n");
}

__global__
void calc_answer(const int *matrix, const int *vector, int *answer, long vector_length) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (long i = tid; i < vector_length; i += stride) {
        for (long j = 0; j < vector_length; j++) {
            answer[i] += matrix[i * vector_length + j] * vector[j];
        }
    }
}

void save_answer(FILE *output_file, const int *answer, long answer_length) {
    if (DEBUG) printf("save_answer:\n");
    fprintf(output_file, "result:\n");

    for (long i = 0; i < answer_length; i++) {
        fprintf(output_file, "%d ", answer[i]);
        if (DEBUG) printf("%d ", answer[i]);
    }
    fprintf(output_file, "\n");
    if (DEBUG) printf("\n\n");
}
